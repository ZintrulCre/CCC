
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <chrono>
#include <thread>
#include <string>

using namespace std;
constexpr int magic_number = 12345;

__global__ void Add(int n, int *cuda_x)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        cuda_x[i] += cuda_x[i];
}

int main(void)
{
    int n = 1<<30;
    int64_t byte_size = n * sizeof(int);
    int *x;
    x = (int*)malloc(byte_size);
    for (int i = 0; i < n; ++i)
        x[i] = magic_number;
    
    int *cuda_x;
    hipMalloc((void**)&cuda_x, byte_size);

    // copy from host to device
    hipMemcpy(cuda_x, x, byte_size, hipMemcpyHostToDevice);

    std::chrono::steady_clock::time_point time_begin = std::chrono::steady_clock::now();
    Add<<<1, 1024>>>(n, cuda_x);
    hipDeviceSynchronize();
    std::chrono::steady_clock::time_point time_end = std::chrono::steady_clock::now();
    
    // copy from device to host
    hipMemcpy(x, cuda_x, byte_size, hipMemcpyDeviceToHost);

    // check result
    bool result{ true };
    for (uint32_t i = 0; i < n; ++i)
        result = (result && (x[i] == magic_number + magic_number));
    string result_str = (result ? "true" : "false");

    cout << "result: " << result_str << endl;
    cout << "time: " << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_begin).count() << " ms" << endl;
    free(x);
    hipFree(cuda_x);
    return 0;
}